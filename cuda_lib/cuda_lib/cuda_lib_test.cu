#include "hip/hip_runtime.h"

#include "cuda_lib_test.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addKernel2(int4 *c, const int4 *a, const int4 *b)
{
	int i = threadIdx.x;
	c[i].x = a[i].x + b[i].x;
	c[i].y = a[i].y + b[i].y;
	c[i].z = a[i].z + b[i].z;
	c[i].w = a[i].w + b[i].w;
}

__global__ void SingleLoop()
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
}

//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}


std::stringstream CudaSingleLoop(dim3 &grid, dim3 &block)
{
	std::stringstream ss;
	SingleLoop<<<grid, block>>>(&ss);
	return ss;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size, std::string *error_message)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
		*error_message = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
		*error_message = "hipMalloc failed!";
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
		*error_message = "hipMalloc failed!";
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
		*error_message = "hipMalloc failed!";
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		*error_message = "hipMemcpy failed!";
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		*error_message = "hipMemcpy failed!";
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
		*error_message = "addKernel launch failed: " + std::string(hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
		*error_message = "hipDeviceSynchronize returned error code " + std::to_string(cudaStatus) + " after launching addKernel!";
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		*error_message = "hipMemcpy failed!";
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda2(int4 *c, const int4 *a, const int4 *b, std::string* error_message)
{
	int4 *dev_a = 0;
	int4 *dev_b = 0;
	int4 *dev_c = 0;
	const unsigned int size = 1;
	hipError_t cuda_status;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		*error_message = "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cuda_status = hipMalloc((void**)&dev_c, size * sizeof(int4));
	if (cuda_status != hipSuccess) {
		*error_message = "hipMalloc failed!";
		goto Error;
	}

	cuda_status = hipMalloc((void**)&dev_a, size * sizeof(int4));
	if (cuda_status != hipSuccess) {
		*error_message = "hipMalloc failed!";
		goto Error;
	}

	cuda_status = hipMalloc((void**)&dev_b, size * sizeof(int4));
	if (cuda_status != hipSuccess) {
		*error_message = "hipMalloc failed!";
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cuda_status = hipMemcpy(dev_a, a, size * sizeof(int4), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		*error_message = "hipMemcpy failed!";
		goto Error;
	}

	cuda_status = hipMemcpy(dev_b, b, size * sizeof(int4), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		*error_message = "hipMemcpy failed!";
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel2 <<<1, size>>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		*error_message = "addKernel launch failed: " + std::string(hipGetErrorString(cuda_status));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		*error_message = "hipDeviceSynchronize returned error code " + std::to_string(cuda_status) + " after launching addKernel!";
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cuda_status = hipMemcpy(c, dev_c, size * sizeof(int4), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		*error_message = "hipMemcpy failed!";
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cuda_status;
}